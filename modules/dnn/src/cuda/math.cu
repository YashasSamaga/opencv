#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>

#include "../cuda4dnn/csl/math.hpp"
#include "../cuda4dnn/csl/kernel_utils.hpp"

namespace cv { namespace dnn { namespace cuda4dnn { namespace csl  { namespace kernels {

    namespace detail {
        template <class T> __device__ T abs(T val);
        template <> __device__ float abs(float val) { return fabsf(val); }
        template <> __device__ double abs(double val) { return fabs(val); }

        template <class T> __device__ T exp(T val);
        template <> __device__ float exp(float val) { return expf(val); }
        template <> __device__ double exp(double val) { return exp(val); }

        template <class T> __device__ T max(T x, T y);
        template <> __device__ float max(float x, float y) { return fmaxf(x, y); }
        template <> __device__ double max(double x, double y) { return fmax(x, y); }

        template <class T> __device__ T min(T x, T y);
        template <> __device__ float min(float x, float y) { return fminf(x, y); }
        template <> __device__ double min(double x, double y) { return fmin(x, y); }

        template <class T> __device__ T log(T val);
        template <> __device__ float log(float val) { return logf(val); }
        template <> __device__ double log(double val) { return log(val); }

        template <class T> __device__ T tanh(T val);
        template <> __device__ float tanh(float val) { return tanhf(val); }
        template <> __device__ double tanh(double val) { return tanh(val); }

        template <class T> __device__ T pow(T val, T exp);
        template <> __device__ float pow(float val, float exp) { return powf(val, exp); }
        template <> __device__ double pow(double val, double exp) { return pow(val, exp); }

        template <class T>
        __device__ T sigmoid(T val) { return T(1) / (1 + exp(-val)); }
    }

    namespace raw {
        template <class T>
        __global__ void abs(view<T> src, span<T> dest) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::abs;
                dest[i] = abs(src[i]);
            }
        }

        template <class T>
        __global__ void tanh(view<T> src, span<T> dest) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::tanh;
                dest[i] = tanh(src[i]);
            }
        }

        template <class T>
        __global__ void sigmoid(view<T> src, span<T> dest) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::sigmoid;
                dest[i] = sigmoid(src[i]);
            }
        }

        template <class T>
        __global__ void bnll(view<T> src, span<T> dest) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::log;
                using detail::exp;
                dest[i] = log(1 + exp(-src[i]));
            }
        }

        template <class T>
        __global__ void elu(view<T> src, span<T> dest) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::exp;
                dest[i] = src[i] >= 0 ? src[i] : (exp(src[i]) - 1);
            }
        }

        template <class T>
        __global__ void relu(view<T> src, span<T> dest, T slope) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                dest[i] = src[i] >= 0.0 ? src[i] : slope * src[i];
            }
        }

        template <class T>
        __global__ void clipped_relu(view<T> src, span<T> dest, T ceiling, T floor) {
            assert(src.size() >= dest.size());
            assert(floor <= ceiling);
            for (auto i : grid_stride_range(dest.size())) {
                using detail::max;
                using detail::min;
                dest[i] = min(max(src[i], floor), ceiling);
            }
        }

        template <class T>
        __global__ void power(view<T> src, span<T> dest, T exp, T scale, T shift) {
            assert(src.size() >= dest.size());
            for (auto i : grid_stride_range(dest.size())) {
                using detail::pow;
                dest[i] = pow(shift + scale * src[i], exp);
            }
        }
    }

    template <class T>
    void abs(const Stream& stream, span<T> dest, view<T> src) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::abs<T>, 0, stream);
        launch_kernel(raw::abs<T>, policy, src, dest);
    }

    template void abs<float>(const Stream& stream, span<float> dest, view<float> src);
    template void abs<double>(const Stream& stream, span<double> dest, view<double> src);

    template <class T>
    void tanh(const Stream& stream, span<T> dest, view<T> src) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::tanh<T>, 0, stream);
        launch_kernel(raw::tanh<T>, policy, src, dest);
    }

    template void tanh<float>(const Stream& stream, span<float> dest, view<float> src);
    template void tanh<double>(const Stream& stream, span<double> dest, view<double> src);

    template <class T>
    void sigmoid(const Stream& stream, span<T> dest, view<T> src) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::sigmoid<T>, 0, stream);
        launch_kernel(raw::sigmoid<T>, policy, src, dest);
    }

    template void sigmoid<float>(const Stream& stream, span<float> dest, view<float> src);
    template void sigmoid<double>(const Stream& stream, span<double> dest, view<double> src);

    template <class T>
    void bnll(const Stream& stream, span<T> dest, view<T> src) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::bnll<T>, 0, stream);
        launch_kernel(raw::bnll<T>, policy, src, dest);
    }

    template void bnll<float>(const Stream& stream, span<float> dest, view<float> src);
    template void bnll<double>(const Stream& stream, span<double> dest, view<double> src);

    template <class T>
    void elu(const Stream& stream, span<T> dest, view<T> src) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::elu<T>, 0, stream);
        launch_kernel(raw::elu<T>, policy, src, dest);
    }

    template void elu<float>(const Stream& stream, span<float> dest, view<float> src);
    template void elu<double>(const Stream& stream, span<double> dest, view<double> src);

    template <class T>
    void relu(const Stream& stream, span<T> dest, view<T> src, T slope) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::relu<T>, 0, stream);
        launch_kernel(raw::relu<T>, policy, src, dest, slope);
    }

    template void relu<float>(const Stream& stream, span<float> dest, view<float> src, float slope);
    template void relu<double>(const Stream& stream, span<double> dest, view<double> src, double slope);

    template <class T>
    void clipped_relu(const Stream& stream, span<T> dest, view<T> src, T floor, T ceiling) {
        CV_Assert(src.size() >= dest.size());
        CV_Assert(floor <= ceiling);

        auto policy = make_optimal_policy(raw::clipped_relu<T>, 0, stream);
        launch_kernel(raw::clipped_relu<T>, policy, src, dest, floor, ceiling);
    }

    template void clipped_relu<float>(const Stream& stream, span<float> dest, view<float> src, float floor, float ceiling);
    template void clipped_relu<double>(const Stream& stream, span<double> dest, view<double> src, double floor, double ceiling);

    template <class T>
    void power(const Stream& stream, span<T> dest, view<T> src, T exp, T scale, T shift) {
        CV_Assert(src.size() >= dest.size());

        auto policy = make_optimal_policy(raw::power<T>, 0, stream);
        launch_kernel(raw::power<T>, policy, src, dest, exp, scale, shift);
    }

    template void power<float>(const Stream& stream, span<float> dest, view<float> src, float exp, float scale, float shift);
    template void power<double>(const Stream& stream, span<double> dest, view<double> src, double exp, double scale, double shift);

}}}}} /*  cv::dnn::cuda4dnn::csl::kernels */
